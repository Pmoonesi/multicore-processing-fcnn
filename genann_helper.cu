#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include "genann.h"

// CUDA runtime
#include <hip/hip_runtime.h>
#include <>

hipError_t cudaStatus;

__global__ void
matrixMultiply(double *d_output, double *d_weights, int current_layer, int next_layer)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int length = gridDim.x * blockDim.x;
	
	while (i < next_layer) {
		double *start_w = d_weights + (current_layer + 1) * i;
		double sum = *start_w++ * -1.0;
		for (int j = 0; j < current_layer; j++)
			sum += *start_w++ * *d_output++;
		*(d_output + i) = sum;
		i += length;
	}
}

extern "C" void cudaFeedForward(genann *ann) {
	double *d_output, *d_weights;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		printf("hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
	}

	cudaStatus = hipMalloc((void**)&d_weights, ann->total_weights * sizeof(double));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
	}

	cudaStatus = hipMalloc((void**)&d_output, ann->total_neurons * sizeof(double));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc failed!");
	}

	cudaStatus = hipMemcpy(d_weights, ann->weight, ann->total_weights * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
	}

	cudaStatus = hipMemcpy(d_output, ann->output, ann->total_neurons * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
	}

	if (!ann->hidden_layers) {
		matrixMultiply << <1, 512 >> > (d_output, d_weights, ann->inputs, ann->outputs);
	}
	else {
		double* d_o = d_output;
		double* d_w = d_weights;
		int i;

		/* Figure input layer */
		matrixMultiply << <1, 512 >> > (d_o, d_w, ann->inputs, ann->hidden);
		d_o += ann->inputs;
		d_w += (ann->inputs + 1) * ann->hidden;

		/* Figure hidden layers, if any. */
		for (i = 1; i < ann->hidden_layers; i++) {
			matrixMultiply << <1, 512 >> > (d_o, d_w, ann->hidden, ann->hidden);
			d_o += ann->hidden;
			d_w += (ann->hidden + 1) * ann->hidden;
		}

		/* Figure output layer. */
		matrixMultiply << <1, 512 >> > (d_o, d_w, ann->hidden, ann->outputs);
		d_o += ann->hidden;
		d_w += (ann->hidden + 1) * ann->outputs;

		// making sure we've been through all weights an neurons
		d_o += ann->outputs;
		assert(d_w - d_weights == ann->total_weights);
		assert(d_o - d_output == ann->total_neurons);
	}

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		printf("hipDeviceSynchronize returned error code %d after launching addKernel!\n",
			cudaStatus);
	}

	cudaStatus = hipMemcpy(ann->output, d_output, ann->total_neurons * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy failed!");
	}

	hipFree(d_output);
	hipFree(d_weights);
}